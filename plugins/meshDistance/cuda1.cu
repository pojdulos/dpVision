#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "UI.h"

// Funkcja wykonywana r�wnolegle na wielu danych
// przez jednostki strumieniowe GPU
__global__ void square_array(float *a, int N)
{
	// Obliczamy numer w�tku na podstawie numeru bloku,
	// numeru w�tku w bloku i ilo�ci w�tk�w w bloku
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// Warunek, aby nie wychodzi� poza zakres tablicy,
	// kt�r� b�dziemy przetwarza� oraz w�a�ciwe obliczenie
	// czyli obliczenie kwadratu danego pola, za kt�re
	// odpowiedzialny jest dany w�tek
	if (idx<N) a[idx] = a[idx] * a[idx];
}

// Funkcja wywo�ywana na CPU
void cuda_test()
{
	float *a_h, *a_d;  // Wska�niki do wykorzystywanych tablic
	const int N = 100;  // D�ugo�� tablic
	size_t size = N * sizeof(float); // Rozmiar tablic w bajtach

	a_h = (float *)malloc(size);        // Alokacja pami�ci przez CPU
	hipMalloc((void **)&a_d, size);   // Alokacja pami�ci przez GPU

									   // Inicjalizacja danych przez CPU
	for (int i = 0; i<N; i++) a_h[i] = (float)i;
	// Skopiowanie danych do GPU
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

	// Ustalenie rozmiaru bloku
	int block_size = 4;
	// Obliczenie liczby blok�w koniczecznych do przetworzenia ca�ej tablicy
	int n_blocks = N / block_size + (N%block_size == 0 ? 0 : 1);

	// Specjalna sk�adnia wywo�ania podprogramu dla GPU z podaniem
	// nazwy funkcji, liczby i rozmiaru bloku oraz parametr�w funkcji
	square_array << < n_blocks, block_size >> > (a_d, N);
	// Pobranie danych wynikowych z GPU do CPU
	hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	// Wypisanie wynik�w
	for (int i = 0; i<N; i++) UI::STATUSBAR::printf("%d %f\n", i, a_h[i]);
	// Zwalnianie
	free(a_h); hipFree(a_d);
}
